#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

#include "payloads.h"

rtBuffer<float4, 2> result_buffer; // RGBA32F

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, theLaunchDim, rtLaunchDim, );
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );

rtDeclareVariable(Matrix3x3, view, , );
rtDeclareVariable(float3, origin, , );
rtDeclareVariable(float, fov, , );


rtDeclareVariable(rtObject, sysTopObject, , );

RT_PROGRAM void color_raygen()
{

    RayData pyld;
    pyld.origin = origin;
    pyld.radiance = make_float3(0.0);

    const float2 pixel = make_float2(theLaunchIndex);
    const float2 fragment = pixel + make_float2(0.5f);
    const float2 screen = make_float2(theLaunchDim);
    const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

    float aspectRatio = screen.x / screen.y;
    float focus = tan(fov/2.0);

    float3 direction = view * normalize(make_float3(-ndc.x * aspectRatio, -ndc.y, focus));

    optix::Ray ray = optix::make_Ray(origin, direction, 0, 0.0f, RT_DEFAULT_MAX);
    rtTrace(sysTopObject, ray, pyld);

    result_buffer[launch_index] = make_float4(pyld.radiance, 1.0f);

}