#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "payloads.h"

rtDeclareVariable(RayData, thePrd, rtPayload, );

RT_PROGRAM void miss() {
  thePrd.radiance = make_float3(0.2f, 0.2f, 0.2f);

}