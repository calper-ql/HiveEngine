#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "payloads.h"

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(RayData, thePrd, rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, varNormal,    attribute NORMAL, );

RT_PROGRAM void chit()
{
  // Transform the (unnormalized) object space normals into world space.
  float3 geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
  float3 normal    = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));

  // Check if the ray hit the geometry on the frontface or the backface.
  // The geometric normal is always defined on the front face of the geometry.
  // In this implementation the coordinate systems are right-handed and the frontface triangle winding is counter-clockwise (matching OpenGL).

  // If theRay.direction and geometric normal are in the same hemisphere we're looking at a backface.
  if (0.0f < optix::dot(theRay.direction, geoNormal))
  {
    // Flip the shading normal to the backface, because only that is used below.
    // (See later examples for more intricate handling of the frontface condition.)
    normal = -normal;
  }

  // Visualize the resulting world space normal on the surface we're looking on.
  // Transform the normal components from [-1.0f, 1.0f] to the range [0.0f, 1.0f] to get colors for negative values.
  thePrd.radiance = normal * 0.5f + 0.5f;
}